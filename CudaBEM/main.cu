﻿#include <iostream>
#include <chrono>
#include <vector>

#include "hip/hip_runtime.h"
#include "Point.h"
#include "Element.h"
#include "Input.h"
#include "Gauss.cuh"
#include "Calculator.cuh"


void integrate_index(size_t index, size_t element_count, Point* points, Element* elements, double* q, double* p, Vector Y, double* result)
{
	if (index >= element_count)
		return;

	Element el = elements[index];
	Point A = points[el.v1];
	Point B = points[el.v2];
	Point C = points[el.v3];

	Vector v1 = B - A;
	Vector v2 = C - A;
	Vector normal = v1.Cross(v2).Normalize();

	Vector Q(q[el.q1], q[el.q2], q[el.q3]);
	double DuDn = p[el.p];

	Vector X;

	for (int i = 0; i < 66; i++)
	{
		double ksi = p1h[i];
		double etta = p2h[i];
		double weight = wh[i];

		Point L(1 - ksi - etta, ksi, etta);

		double U = L * Q;
		X.x = L * Vector(A.x, B.x, C.x);
		X.y = L * Vector(A.y, B.y, C.y);
		X.z = L * Vector(A.z, B.z, C.z);

		double f = F1(X, Y, DuDn) + F2(X, Y, normal, U);

		result[index] += 0.25 * weight * f;
	}

	result[index] *= v1.Cross(v2).Norm();
}

double Integrate(vector<Point>& points, vector<Element>& elements, vector<double>& q, vector<double>& p, Vector& Y)
{
	double result = 0;

	double* result_array = new double[elements.size()];
	int index = 0;
	for (auto& el : elements)
	{
		Point A = points[el.v1];
		Point B = points[el.v2];
		Point C = points[el.v3];

		Vector v1 = B - A;
		Vector v2 = C - A;

		Vector normal = v1.Cross(v2).Normalize();

		Vector Q(q[el.q1], q[el.q2], q[el.q3]);
		double DuDn = p[el.p];

		Vector X;

		double res_i = 0;
		for (int i = 0; i < 66; i++)
		{
			double ksi = p1h[i];
			double etta = p2h[i];
			double weight = wh[i];

			Point L(1 - ksi - etta, ksi, etta);

			double U = L * Q;
			X.x = L * Vector(A.x, B.x, C.x);
			X.y = L * Vector(A.y, B.y, C.y);
			X.z = L * Vector(A.z, B.z, C.z);

			double f = F1(X, Y, DuDn) + F2(X, Y, normal, U);

			res_i += 0.25 * weight * f;
		}

		res_i *= v1.Cross(v2).Norm();

		result_array[index++] = res_i;
	}


	for (int i = 0; i < elements.size(); i++)
		result += result_array[i];

	return result;
}

double IntegrateIndex(vector<Point>& points, vector<Element>& elements, vector<double>& q, vector<double>& p, Vector& Y)
{
	size_t size = elements.size();

	double* host_result = new double[size];
	memset(host_result, 0, size * sizeof(double));

	for (int i = 0; i < size; i++)
		integrate_index(i, elements.size(), points.data(), elements.data(), q.data(), p.data(), Y, host_result);

	double res = 0.0;
	for (int i = 0; i < size; i++)
		res += host_result[i];

	return res;
}

int main(void)
{
	std::vector<Point> points;
	std::vector<double> q;
	std::vector<double> p;
	std::vector<Element> elements;

	Input("test/points.txt", points, "test/weights.txt", q, p, "test/triangles.txt", elements);

	Point target;

	ifstream in("test/target.txt");
	in >> target.x >> target.y >> target.z;
	in.close();

	auto start = std::chrono::steady_clock::now();
	double result = Integrate(points, elements, q, p, target);
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	std::cout << "Integrate. Elapsed time: " << elapsed_seconds.count() << "s\n";

	calculator* calc = nullptr;
	create_calculator(points, elements, q, p, &calc);

	start = std::chrono::steady_clock::now();
	double resultCuda = calculate_value(calc, target.x, target.y, target.z);
	end = std::chrono::steady_clock::now();
	elapsed_seconds = end - start;
	std::cout << "IntegrateCuda. Elapsed time: " << elapsed_seconds.count() << "s\n";

	start = std::chrono::steady_clock::now();
	double resultIndex = IntegrateIndex(points, elements, q, p, target);
	end = std::chrono::steady_clock::now();
	elapsed_seconds = end - start;
	std::cout << "IntegrateIndex. Elapsed time: " << elapsed_seconds.count() << "s\n";

	return 0;
}